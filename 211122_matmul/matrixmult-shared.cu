#include "hip/hip_runtime.h"
﻿

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include<string.h>
#include<malloc.h>
#include<windows.h>
#include<time.h>
#include<Winbase.h>
#include<cstring>
#include<cstdio>
#include<math.h>
#include<io.h>
#include<fcntl.h>
#include<sys/stat.h>
#include<chrono>
#include<crt/hip/device_functions.h>

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do{\
	(x);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess!=e){\
		printf("cuda failure \"%s\" at %s:%d\n", \
			hipGetErrorString(e),\
			__FILE__, __LINE__);\
		exit(1);\
	}\
}while(0)
#endif

using namespace std;
using namespace chrono;
typedef duration<long long, nano> nanoseconds;
typedef duration<long long, micro> microsecons;
typedef duration<long long, milli> milliseconds;

#define WIDTH 8
#define TILE_WIDTH 2
#define WARPSIZE TILE_WIDTH
#define GRIDSIZE ((WIDTH/TILE_WIDTH)*(WIDTH/TILE_WIDTH))
#define BLOCKSIZE (TILE_WIDTH*TILE_WIDTH)
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE)

void genDataA(unsigned* ptr, int size)
{
	unsigned int gy = 0, gx = 0;
	for (unsigned int gy = 0; gy < WIDTH; gy++) {
		for (unsigned int gx = 0; gx < WIDTH; gx++) {
			*ptr++ = gy * WIDTH + gx;
		}
	}
}
void genDataB(unsigned* ptr, int size)
{
	unsigned int gy = 0, gx = 0;
	for (unsigned int gy = 0; gy < WIDTH; gy++) {
		for (unsigned int gx = 0; gx < WIDTH; gx++) {
			*ptr++ = (gx * WIDTH + gy);
		}
	}
}
void printData(unsigned* ptr, int size)
{
	for (unsigned int gy = 0; gy < WIDTH; gy++) {
		for (unsigned int gx = 0; gx < WIDTH; gx++) {
			printf("%-3d ", ptr[gy * WIDTH + gx]);
		}
		printf("\n");
	}
}
//width에 보내줄 변수는 for문에서 k의 종결조건으로 사용할 A의 총 열의 개수 혹은 B의 총 행의 개수
//width는 WIDTH
void matmulti_host(const unsigned* g_A, const unsigned* g_B, unsigned* g_C, const int width) {
	for (register int gy = 0; gy < width; gy++) {
		for (register int gx = 0; gx < width; gx++) {
			register unsigned sum = 0;
			for (register int k = 0; k < width; k++) {
				sum += g_A[gy * width + k] * g_B[k * width + gx];
			}
			g_C[gy * width + gx] = sum;
		}
	}
}
//width에 보내줄 변수는 for문에서 k의 종결조건으로 사용할 A의 총 열의 개수 혹은 B의 총 행의 개수
//width는 WIDTH
__global__ void matmulti_shared(const unsigned* g_A, const unsigned* g_B, unsigned* g_C, const int width)
{
	__shared__ unsigned s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ unsigned s_B[TILE_WIDTH][TILE_WIDTH];
	int by = blockIdx.y, bx = blockIdx.x;
	int ty = threadIdx.y, tx = threadIdx.x;
	int gy = by * TILE_WIDTH + ty;
	int gx = bx * TILE_WIDTH + tx;
	unsigned sum = 0;
	for (register int m = 0; m < width / TILE_WIDTH; m++) {
		//global memory에서 shared memory로 가져올 때 gy,gx는 고정해두고
		s_A[ty][tx] = g_A[gy * width + (m * TILE_WIDTH + tx)];
		s_B[ty][tx] = g_B[(m * TILE_WIDTH + ty) * width + gx];
		__syncthreads();
		//k는 shared memory들로 g_C의 성분을 구하기 위한 것이므로
		//위의 설명에 따라 k는 TILE_WIDTH가 종결조건입니다
		for (register int k = 0; k < TILE_WIDTH; k++) {
			sum += s_A[ty][k] * s_B[k][tx];
		}
		__syncthreads();
	}
	g_C[gy * width + gx] = sum;
}
__host__ int main(void)
{
	//바로 아래 두 줄은 QueryPerformance로 CUDA이벤트 쓸때는 
	//float형으로 아래의 메모리할당을 해주어야 오류가 안 나길래 적었습니다
	float* pSource = NULL;
	float* pResult = NULL;
	//시간변수선언
	long long cntStart, cntEnd, freq;
	QueryPerformanceFrequency((LARGE_INTEGER*)(&freq));
	pSource = (float*)malloc(sizeof(float) * TOTALSIZE);
	pResult = (float*)malloc(sizeof(float) * TOTALSIZE);

	//host변수 선언 및 초기화
	unsigned* pA = NULL;
	unsigned* pB = NULL;
	unsigned* pC = NULL;
	pA = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	pB = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	pC = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
	genDataA(pA, TOTALSIZE);
	genDataB(pB, TOTALSIZE);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	printf("elasped time : %f usec\n", (double)(cntEnd - cntStart) * 10000.0 / (double)(freq));
	printf("g_A component;\n");
	printData(pA, TOTALSIZE);
	printf("g_B component;\n");
	printData(pB, TOTALSIZE);
	//device변수 선언
	unsigned* pADev = NULL;
	unsigned* pBDev = NULL;
	unsigned* pCDev = NULL;
	//hipMalloc,hipMemset, hipMemcpy:H->D
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
	CUDA_CHECK(hipMalloc((void**)&pADev, sizeof(unsigned) * TOTALSIZE));//WIDTH*WIDTH
	CUDA_CHECK(hipMemset(pADev, 0, sizeof(unsigned) * TOTALSIZE));
	CUDA_CHECK(hipMemcpy(pADev, pA, sizeof(unsigned) * TOTALSIZE, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMalloc((void**)&pBDev, sizeof(unsigned) * TOTALSIZE));//WIDTH*WIDTH
	CUDA_CHECK(hipMemset(pBDev, 0, sizeof(unsigned) * TOTALSIZE));
	CUDA_CHECK(hipMemcpy(pBDev, pB, sizeof(unsigned) * TOTALSIZE, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMalloc((void**)&pCDev, sizeof(unsigned) * TOTALSIZE));//WIDTH*WIDTH
	CUDA_CHECK(hipMemset(pCDev, 0, sizeof(unsigned) * TOTALSIZE));
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	printf("elapsed time for 디바이스변수 초기화= %f msec\n", (double)(cntEnd - cntStart) * 100000.0 / (double)(freq));
	//cuda kernel 시간측정시작
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
	dim3 dimGrid(GRIDSIZE, 1, 1);
	dim3 dimBlock(BLOCKSIZE, 1, 1);
	matmulti_shared<<<dimGrid,dimBlock>>>(pADev, pBDev, pCDev, WIDTH);
	//cuda kernel 시간측정 종료
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	CUDA_CHECK(hipPeekAtLastError());
	//hipMemcpy:D->H, host변수 출력
	CUDA_CHECK(hipMemcpy(pC, pCDev, sizeof(unsigned) * TOTALSIZE, hipMemcpyDeviceToHost));
	for (unsigned int y = 0; y < WIDTH; y++) {
		for (unsigned int x = 0; x < WIDTH; x++) {
			printf("g_C[%4d][%4d] = %u\n", y, x, pC[y * WIDTH + x]);
		}
		printf("\n");
	}
	//시간출력
	printf("elapsed time = %f msec\n", (double)(cntEnd - cntStart) * 100000.0 / (double)(freq));
	printData(pC, GRIDSIZE);
	fflush(stdout);
	//host변수 메모리해제
	free(pSource);
	free(pResult);
	free(pA);
	free(pB);
	free(pC);
	//hipFree
	CUDA_CHECK(hipFree(pADev));
	CUDA_CHECK(hipFree(pBDev));
	CUDA_CHECK(hipFree(pCDev));
	//fflush, return 0
	fflush(stdout);
	return 0;
}