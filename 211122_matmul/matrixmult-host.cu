﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include<string.h>
#include<malloc.h>
#include<windows.h>
#include<time.h>
#include<Winbase.h>
#include<cstring>
#include<cstdio>
#include<math.h>
#include<io.h>
#include<fcntl.h>
#include<sys/stat.h>
#include<chrono>
#include<crt/hip/device_functions.h>

#if defined(NDEBUG)
#define CUDA_CHECK(x)	(x)
#else
#define CUDA_CHECK(x)	do{\
	(x);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess!=e){\
		printf("cuda failure \"%s\" at %s:%d\n", \
			hipGetErrorString(e),\
			__FILE__, __LINE__);\
		exit(1);\
	}\
}while(0)
#endif

using namespace std;
using namespace chrono;
typedef duration<long long, nano> nanoseconds;
typedef duration<long long, micro> microsecons;
typedef duration<long long, milli> milliseconds;

#define WIDTH 8
#define TILE_WIDTH 2
#define WARPSIZE TILE_WIDTH
#define GRIDSIZE ((WIDTH/TILE_WIDTH)*(WIDTH/TILE_WIDTH))
#define BLOCKSIZE (TILE_WIDTH*TILE_WIDTH)
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE)

void genDataA(unsigned* ptr, int size)
{
	unsigned int gy = 0, gx = 0;
	for (unsigned int gy = 0; gy < WIDTH; gy++) {
		for (unsigned int gx = 0; gx < WIDTH; gx++) {
			*ptr++ = gy * WIDTH + gx;
		}
	}
}
void genDataB(unsigned* ptr, int size)
{
	unsigned int gy = 0, gx = 0;
	for (unsigned int gy = 0; gy < WIDTH; gy++) {
		for (unsigned int gx = 0; gx < WIDTH; gx++) {
			*ptr++ = (gx * WIDTH + gy);
		}
	}
}
void printData(unsigned* ptr, int size)
{
	for (unsigned int gy = 0; gy < WIDTH; gy++) {
		for (unsigned int gx = 0; gx < WIDTH; gx++) {
			printf("%-7d ", ptr[gy * WIDTH + gx]);
		}
		printf("\n");
	}
}
//width에 보내줄 변수는 for문에서 k의 종결조건으로 사용할 A의 총 열의 개수 혹은 B의 총 행의 개수
//width는 WIDTH
void matmulti_host(const unsigned* g_A, const unsigned* g_B, unsigned* g_C, const int width) {
	for (register int gy = 0; gy < width; gy++) {
		for (register int gx = 0; gx < width; gx++) {
			register unsigned sum = 0;
			for (register int k = 0; k < width; k++) {
				sum += g_A[gy * width + k] * g_B[k * width + gx];
			}
			g_C[gy * width + gx] = sum;
		}
	}
}

__host__ int main(void)
{
	//바로 아래 두 줄은 QueryPerformance로 CUDA이벤트 쓸때는 
	//float형으로 아래의 메모리할당을 해주어야 오류가 안 나길래 적었습니다
	float* pSource = NULL;
	float* pResult = NULL;
	long long cntStart, cntEnd, freq;
	QueryPerformanceFrequency((LARGE_INTEGER*)(&freq));
	pSource = (float*)malloc(sizeof(float) * TOTALSIZE);
	pResult = (float*)malloc(sizeof(float) * TOTALSIZE);

	//host변수 선언 및 초기화
	unsigned* pA = NULL;
	unsigned* pB = NULL;
	unsigned* pC = NULL;
	pA = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	pB = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	pC = (unsigned*)malloc(sizeof(unsigned) * TOTALSIZE);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
	genDataA(pA, TOTALSIZE);
	genDataB(pB, TOTALSIZE);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	printf("elasped time : %f usec\n", (double)(cntEnd - cntStart) * 10000.0 / (double)(freq));
	printf("g_A component;\n");
	printData(pA, TOTALSIZE);
	printf("g_B component;\n");
	printData(pB, TOTALSIZE);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
	matmulti_host(pA, pB, pC, WIDTH);
	QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));
	printf("elapsed time = %f msec\n", (double)(cntEnd - cntStart) * 100000.0 / (double)freq);
	printData(pC, GRIDSIZE);
	//마지막에 host변수 메모리해제코드
	free(pSource);
	free(pResult);
	free(pA);
	free(pB);
	free(pC);
}